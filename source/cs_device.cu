#include "hip/hip_runtime.h"
/*
 * CudaSift library
 *
 * Copyright (C) 2007-2015 Marten Bjorkman <celle@nada.kth.se>
 * Copyright (C) 2015 Siim Meerits <siim@yutani.ee>
 */

// Internal
#include "cs_device.h"

// C++
#include <cassert>

// CUDA error checking macro
static void cuda_safecall(hipError_t ret, const char* file, const char* func, const int line)
{
    if (ret != hipSuccess) {
        printf("[cudasift] CUDA error: %s! (%s:%s:%d)\n", hipGetErrorString(ret), file, func, line);
    }
}
#define CUDA_SAFECALL(ret) cuda_safecall(ret, __FILE__, __func__, __LINE__)

inline int iDivUp(int a, int b) { return (a%b != 0) ? (a/b + 1) : (a/b); }
inline int iDivDown(int a, int b) { return a/b; }
inline int iAlignUp(int a, int b) { return (a%b != 0) ?  (a - a%b + b) : a; }
inline int iAlignDown(int a, int b) { return a - a%b; }

namespace cudasift
{

// FindPoints, FindPointsMulti
__constant__ float d_Threshold[2];
__constant__ float d_Scales[8];
__constant__ float d_Factor;
__constant__ float d_EdgeLimit;
__constant__ int d_MaxNumPoints;
__device__ unsigned int d_PointCounter[1];

// ExtractSiftDescriptors
texture<float, 2, hipReadModeElementType> tex;

// ...
__device__ __constant__ float d_Kernel[12 * 16]; // NOTE: Maximum radius

// Set and get functions
hipError_t set_threshold(float a, float b)
{
    float r[] = { a, b };
    return hipMemcpyToSymbol(HIP_SYMBOL(d_Threshold), &r, 2 * sizeof(float));
}

hipError_t set_scales(float* scales, size_t size)
{
    assert(size <= 8);
    return hipMemcpyToSymbol(HIP_SYMBOL(d_Scales), scales, size * sizeof(float));
}

hipError_t set_factor(float factor)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(d_Factor), &factor, sizeof(float));
}

hipError_t set_edge_limit(float edge_limit)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(d_EdgeLimit), &edge_limit, sizeof(float));
}

hipError_t set_max_points(int max_points)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(d_MaxNumPoints), &max_points, sizeof(int));
}

hipError_t set_point_counter(unsigned int points)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(d_PointCounter), &points, sizeof(unsigned int));
}

hipError_t get_point_counter(unsigned int& points)
{
    return hipMemcpyFromSymbol(&points, HIP_SYMBOL(d_PointCounter), sizeof(unsigned int));
}

hipError_t set_kernel(float* kernel, size_t size)
{
    assert(size <= 12 * 16);
    return hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), kernel, size * sizeof(float));
}

void cpu_scale_down(cv::cuda::GpuMat& dst, cv::cuda::GpuMat& src, float variance)
{
    assert(src.cols > 0 && src.rows > 0 && src.type() == CV_32FC1);

    cv::cuda::PtrStepSzf gpu_dst = dst;
    cv::cuda::PtrStepSzf gpu_src = src;

    // Kernel
    float kernel[5];
    float kernelSum = 0.0f;
    for (int j=0;j<5;j++) {
        kernel[j] = (float)expf(-(double)(j-2)*(j-2)/2.0/variance);
        kernelSum += kernel[j];
    }
    for (int j=0;j<5;j++)
        kernel[j] /= kernelSum;
    CUDA_SAFECALL(set_kernel(kernel, 5));

    // Scale down
    dim3 blocks(iDivUp(gpu_src.cols, SCALEDOWN_W), iDivUp(gpu_src.rows, SCALEDOWN_H));
    dim3 threads(SCALEDOWN_W + WARP_SIZE + 2);
    gpu_scale_down<<<blocks, threads>>>(gpu_dst.ptr(), gpu_src.ptr(), gpu_src.cols, gpu_src.step / sizeof(float), gpu_src.rows, gpu_dst.step / sizeof(float));
    CUDA_SAFECALL(hipGetLastError());
    CUDA_SAFECALL(hipDeviceSynchronize());
}

__global__ void gpu_scale_down(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
    __shared__ float inrow[SCALEDOWN_W+4];
    __shared__ float brow[5*(SCALEDOWN_W/2)];
    __shared__ int yRead[SCALEDOWN_H+4], yWrite[SCALEDOWN_H+4];

    #define dx2 (SCALEDOWN_W/2)
    const int tx = threadIdx.x;
    const int tx0 = tx + 0*dx2;
    const int tx1 = tx + 1*dx2;
    const int tx2 = tx + 2*dx2;
    const int tx3 = tx + 3*dx2;
    const int tx4 = tx + 4*dx2;
    const int xStart = blockIdx.x*SCALEDOWN_W;
    const int yStart = blockIdx.y*SCALEDOWN_H;
    const int xWrite = xStart/2 + tx;
    const float *k = d_Kernel;

    if (tx<SCALEDOWN_H+4) {
        int y = yStart + tx - 1;
        y = (y<0 ? 0 : y);
        y = (y>=height ? height-1 : y);
        yRead[tx] = y*pitch;
        yWrite[tx] = (yStart + tx - 4)/2 * newpitch;
    }

    __syncthreads();

    int xRead = xStart + tx - WARP_SIZE;
    xRead = (xRead<0 ? 0 : xRead);
    xRead = (xRead>=width ? width-1 : xRead);

    for (int dy=0;dy<SCALEDOWN_H+4;dy+=5) {
        if (tx>=WARP_SIZE-2)
            inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+0] + xRead];
        __syncthreads();
        if (tx<dx2)
            brow[tx0] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
        __syncthreads();
        if (tx<dx2 && dy>=4 && !(dy&1))
            d_Result[yWrite[dy+0] + xWrite] = k[2]*brow[tx2] + k[0]*(brow[tx0]+brow[tx4]) + k[1]*(brow[tx1]+brow[tx3]);

        if (dy<(SCALEDOWN_H+3)) {
            if (tx>=WARP_SIZE-2)
                inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+1] + xRead];
            __syncthreads();
            if (tx<dx2)
                brow[tx1] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
            __syncthreads();
            if (tx<dx2 && dy>=3 && (dy&1))
                d_Result[yWrite[dy+1] + xWrite] = k[2]*brow[tx3] + k[0]*(brow[tx1]+brow[tx0]) + k[1]*(brow[tx2]+brow[tx4]);
        }

        if (dy<(SCALEDOWN_H+2)) {
            if (tx>=WARP_SIZE-2)
                inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+2] + xRead];
            __syncthreads();
            if (tx<dx2)
                brow[tx2] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
            __syncthreads();
            if (tx<dx2 && dy>=2 && !(dy&1))
                d_Result[yWrite[dy+2] + xWrite] = k[2]*brow[tx4] + k[0]*(brow[tx2]+brow[tx1]) + k[1]*(brow[tx3]+brow[tx0]);
        }

        if (dy<(SCALEDOWN_H+1)) {
            if (tx>=WARP_SIZE-2)
                inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+3] + xRead];
            __syncthreads();
            if (tx<dx2)
                brow[tx3] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
            __syncthreads();
            if (tx<dx2 && dy>=1 && (dy&1))
                d_Result[yWrite[dy+3] + xWrite] = k[2]*brow[tx0] + k[0]*(brow[tx3]+brow[tx2]) + k[1]*(brow[tx4]+brow[tx1]);
        }

        if (dy<SCALEDOWN_H) {
            if (tx>=WARP_SIZE-2)
                inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+4] + xRead];
            __syncthreads();
            if (tx<dx2)
                brow[tx4] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
            __syncthreads();
            if (tx<dx2 && !(dy&1))
                d_Result[yWrite[dy+4] + xWrite] = k[2]*brow[tx1] + k[0]*(brow[tx4]+brow[tx3]) + k[1]*(brow[tx0]+brow[tx2]);
        }
        __syncthreads();
    }
}

void cpu_subtract(cv::cuda::GpuMat& dst, cv::cuda::GpuMat& src_a, cv::cuda::GpuMat& src_b)
{
    assert(src_a.cols > 0 && src_a.rows > 0 && src_a.type() == CV_32FC1);
    assert(src_a.cols == src_b.cols && src_a.rows == src_b.rows && src_a.type() == src_b.type());

    dst = cv::cuda::GpuMat(src_a.size(), src_a.type());

    cv::cuda::PtrStepSzf gpu_dst = dst;
    cv::cuda::PtrStepSzf gpu_src_a = src_a;
    cv::cuda::PtrStepSzf gpu_src_b = src_b;

    int w = gpu_dst.cols;
    int p = gpu_dst.step / sizeof(float);
    int h = gpu_dst.rows;

    dim3 blocks(iDivUp(w, SUBTRACT_W), iDivUp(h, SUBTRACT_H));
    dim3 threads(SUBTRACT_W, SUBTRACT_H);
    gpu_subtract<<<blocks, threads>>>(gpu_dst.ptr(), gpu_src_a.ptr(), gpu_src_b.ptr(), w, p, h);
    CUDA_SAFECALL(hipGetLastError());
    CUDA_SAFECALL(hipDeviceSynchronize());
}

__global__ void gpu_subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * SUBTRACT_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACT_H + threadIdx.y;
    int p = y * pitch + x;
    if (x < width && y < height)
        d_Result[p] = d_Data1[p] - d_Data2[p];
    __syncthreads();
}

void cpu_extract_sift_descriptors(
    cv::cuda::GpuMat& image, cv::cuda::GpuMat& sift, cv::cuda::GpuMat& desc,
    int numPts, int maxPts
) {
    // Pointers
    cv::cuda::PtrStepSzf gpu_image = image;
    cv::cuda::PtrStepSzf gpu_sift = sift;
    cv::cuda::PtrStepSzf gpu_desc = desc;

    // Bind texture
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = false;
    size_t offset = 0;
    CUDA_SAFECALL(hipBindTexture2D(&offset, tex, gpu_image.ptr(), tex.channelDesc, gpu_image.cols, gpu_image.rows, gpu_image.step));

    // Calculate descriptors
    dim3 blocks(numPts);
    dim3 threads(16);
    gpu_extract_sift_descriptors<<<blocks, threads>>>(gpu_image.ptr(), gpu_sift.ptr(), gpu_desc.ptr(), maxPts);
    CUDA_SAFECALL(hipGetLastError());
    CUDA_SAFECALL(hipDeviceSynchronize());

    // Unbind texture
    CUDA_SAFECALL(hipUnbindTexture(tex));
}

__global__ void gpu_extract_sift_descriptors(float *g_Data, float *d_sift, float *d_desc, int maxPts)
{
    __shared__ float buffer[NUMDESCBUFS*128];
    __shared__ float gauss[16];
    __shared__ float gradients[256];
    __shared__ float angles[256];

    const int tx = threadIdx.x; // 0 -> 16
    const int bx = blockIdx.x;  // 0 -> numPts

    gauss[tx] = exp(-(tx-7.5f)*(tx-7.5f)/128.0f);

    __syncthreads();

    float theta = 2.0f*3.1415f/360.0f*d_sift[CUDASIFT_POINT_ORIENTATION * maxPts + bx];
    float sina = sinf(theta);           // cosa -sina
    float cosa = cosf(theta);           // sina  cosa
    float scale = 12.0f/16.0f*d_sift[CUDASIFT_POINT_SCALE * maxPts + bx];
    float ssina = scale*sina;
    float scosa = scale*cosa;

    // Compute angles and gradients
    float xpos = d_sift[CUDASIFT_POINT_XPOS * maxPts + bx] + (tx-7.5f)*scosa + 7.5f*ssina;
    float ypos = d_sift[CUDASIFT_POINT_YPOS * maxPts + bx] + (tx-7.5f)*ssina - 7.5f*scosa;

    for (int i=0;i<128*NUMDESCBUFS/16;i++)
        buffer[16*i+tx] = 0.0f;

    for (int y=0;y<16;y++) {
        float dx = tex2D(tex, xpos+cosa, ypos+sina) - tex2D(tex, xpos-cosa, ypos-sina);
        float dy = tex2D(tex, xpos-sina, ypos+cosa) - tex2D(tex, xpos+sina, ypos-cosa);
        gradients[16*y+tx] = gauss[y]*gauss[tx] * sqrtf(dx*dx + dy*dy);
        angles[16*y+tx] = 4.0f/3.1415f*atan2f(dy, dx) + 4.0f;
        xpos -= ssina;
        ypos += scosa;
    }

    __syncthreads();

    if (tx<NUMDESCBUFS) {
        for (int txi=tx;txi<16;txi+=NUMDESCBUFS) {
            int hori = (txi + 2)/4 - 1;
            float horf = (txi - 1.5f)/4.0f - hori;
            float ihorf = 1.0f - horf;
            int veri = -1;
            float verf = 1.0f - 1.5f/4.0f;
            for (int y=0;y<16;y++) {
              int i = 16*y + txi;
              float grad = gradients[i];
              float angf = angles[i];
              int angi = angf;
              int angp = (angi<7 ? angi+1 : 0);
              angf -= angi;
              float iangf = 1.0f - angf;
              float iverf = 1.0f - verf;
              int hist = 8*(4*veri + hori);
              //printf("%d\n", hist);
              int p1 = tx + NUMDESCBUFS*(angi+hist);
              int p2 = tx + NUMDESCBUFS*(angp+hist);
              if (txi>=2) {
                float grad1 = ihorf*grad;
                if (y>=2) {
                    float grad2 = iverf*grad1;
                    buffer[p1+0] += iangf*grad2;
                    buffer[p2+0] +=  angf*grad2;
                }
                if (y<=14) {
                    float grad2 = verf*grad1;
                    buffer[p1+32*NUMDESCBUFS] += iangf*grad2;
                    buffer[p2+32*NUMDESCBUFS] +=  angf*grad2;
                }
              }
              if (txi<=14) {
                  float grad1 = horf*grad;
                  if (y>=2) {
                      float grad2 = iverf*grad1;
                      buffer[p1+8*NUMDESCBUFS] += iangf*grad2;
                      buffer[p2+8*NUMDESCBUFS] +=  angf*grad2;
                  }
                  if (y<=14) {
                      float grad2 = verf*grad1;
                      buffer[p1+40*NUMDESCBUFS] += iangf*grad2;
                      buffer[p2+40*NUMDESCBUFS] +=  angf*grad2;
                  }
              }
              verf += 0.25f;
              if (verf>1.0f) {
                  verf -= 1.0f;
                  veri ++;
              }
            }
        }
    }
    __syncthreads();
    const int t2 = (tx&14)*8;
    const int tx2 = (tx&1);
    for (int i=0;i<16;i++)
        buffer[NUMDESCBUFS*(i+t2)+tx2] += buffer[NUMDESCBUFS*(i+t2)+tx2+2];
    __syncthreads();

    const int t1 = tx*8;
    const int bptr = NUMDESCBUFS*tx + 2;
    buffer[bptr] = 0.0f;
    for (int i=0;i<8;i++) {
        int p = NUMDESCBUFS*(i+t1);
        buffer[p] += buffer[p+1];
        buffer[bptr] += buffer[p]*buffer[p];
    }
    __syncthreads();

    if (tx<8)
        buffer[bptr] += buffer[bptr+8*NUMDESCBUFS];
    __syncthreads();
    if (tx<4)
        buffer[bptr] += buffer[bptr+4*NUMDESCBUFS];
    __syncthreads();
    if (tx<2)
        buffer[bptr] += buffer[bptr+2*NUMDESCBUFS];
    __syncthreads();
    float isum = 1.0f / sqrt(buffer[2] + buffer[NUMDESCBUFS+2]);

    buffer[bptr] = 0.0f;
    for (int i=0;i<8;i++) {
        int p = NUMDESCBUFS*(i+t1);
        buffer[p] = isum*buffer[p];
        if (buffer[p]>0.2f)
            buffer[p] = 0.2f;
        buffer[bptr] += buffer[p]*buffer[p];
    }
    __syncthreads();

    if (tx<8)
        buffer[bptr] += buffer[bptr+8*NUMDESCBUFS];
    __syncthreads();
    if (tx<4)
        buffer[bptr] += buffer[bptr+4*NUMDESCBUFS];
    __syncthreads();
    if (tx<2)
        buffer[bptr] += buffer[bptr+2*NUMDESCBUFS];
    __syncthreads();
    isum = 1.0f / sqrt(buffer[2] + buffer[NUMDESCBUFS+2]);

    for (int i=0;i<8;i++) {
        int p = NUMDESCBUFS*(i+t1);
        d_desc[128*bx+(i+t1)] = isum*buffer[p];
    }
}

unsigned int cpu_find_points(
    cv::cuda::GpuMat& data1, cv::cuda::GpuMat& data2, cv::cuda::GpuMat& data3, cv::cuda::GpuMat& sift,
    float thresh, int maxPts, float edgeLimit, float scale, float factor
) {
    assert(data1.cols > 0 && data1.rows > 0 && data1.type() == CV_32FC1);
    assert(data2.cols > 0 && data2.rows > 0 && data2.type() == CV_32FC1);
    assert(data3.cols > 0 && data3.rows > 0 && data3.type() == CV_32FC1);

    cv::cuda::PtrStepSzf gpu_data1 = data1;
    cv::cuda::PtrStepSzf gpu_data2 = data2;
    cv::cuda::PtrStepSzf gpu_data3 = data3;
    cv::cuda::PtrStepSzf gpu_sift = sift;

    int w = gpu_data1.cols;
    int p = gpu_data1.step / sizeof(float);
    int h = gpu_data1.rows;

    CUDA_SAFECALL(set_threshold(thresh, -thresh));
    CUDA_SAFECALL(set_edge_limit(edgeLimit));
    CUDA_SAFECALL(set_scales(&scale, 1));
    CUDA_SAFECALL(set_factor(factor));
    CUDA_SAFECALL(set_max_points(maxPts));

    CUDA_SAFECALL(set_point_counter(0));

    dim3 blocks(iDivUp(w, MINMAX_W), iDivUp(h, MINMAX_H));
    dim3 threads(MINMAX_W + 2);
    gpu_find_points<<<blocks, threads>>>(gpu_data1.ptr(), gpu_data2.ptr(), gpu_data3.ptr(), gpu_sift.ptr(), w, p, h);
    CUDA_SAFECALL(hipGetLastError());
    CUDA_SAFECALL(hipDeviceSynchronize());

    unsigned int total_points = 0;
    CUDA_SAFECALL(get_point_counter(total_points));
    return total_points;
}

__global__ void gpu_find_points(
    float *d_Data1, float *d_Data2, float *d_Data3, float *d_Sift,
    int width, int pitch, int height
) {
    #define MEMWID (MINMAX_W + 2)
    __shared__ float data1[3*MEMWID], data2[3*MEMWID], data3[3*MEMWID];
    __shared__ float ymin1[MEMWID],   ymin2[MEMWID],   ymin3[MEMWID];
    __shared__ float ymax1[MEMWID],   ymax2[MEMWID],   ymax3[MEMWID];

    const int tx = threadIdx.x;
    const int minx = blockIdx.x*MINMAX_W;
    const int maxx = min(minx + MINMAX_W, width);
    const int xpos = minx + tx;

    int ptr0 = tx;
    int ptr1 = tx;
    int yq = 0;
    for (int y=0;y<MINMAX_H+2;y++) {

        int ypos = MINMAX_H*blockIdx.y + y - 1;
        int yptr = min(max(ypos, 0), height - 1)*pitch;
        int xposr = xpos - 1;
        int ptr2 = yq*MEMWID + tx;

        if (xposr<0) {
            data1[ptr2] = 0;
            data2[ptr2] = 0;
            data3[ptr2] = 0;
        } else if (xposr>=width) {
            data1[ptr2] = 0;
            data2[ptr2] = 0;
            data3[ptr2] = 0;
        } else {
            data1[ptr2] = d_Data1[yptr + xposr];
            data2[ptr2] = d_Data2[yptr + xposr];
            data3[ptr2] = d_Data3[yptr + xposr];
        }
        //__syncthreads();

        if (y>1) {
            float min1 = fminf(fminf(data1[ptr0], data1[ptr1]), data1[ptr2]);
            float min2 = fminf(fminf(data2[ptr0], data2[ptr1]), data2[ptr2]);
            float min3 = fminf(fminf(data3[ptr0], data3[ptr1]), data3[ptr2]);
            float max1 = fmaxf(fmaxf(data1[ptr0], data1[ptr1]), data1[ptr2]);
            float max2 = fmaxf(fmaxf(data2[ptr0], data2[ptr1]), data2[ptr2]);
            float max3 = fmaxf(fmaxf(data3[ptr0], data3[ptr1]), data3[ptr2]);
            ymin1[tx] = min1;
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymin3[tx] = min3;
            ymax1[tx] = max1;
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
            ymax3[tx] = max3;
        }
        //__syncthreads();

        if (y>1) {
            if (tx<MINMAX_W && xpos<maxx) {
                float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx+2]), ymin1[tx+1]), ymin3[tx+1]), data2[ptr0+1]), data2[ptr2+1]);
                minv = fminf(minv, d_Threshold[1]);
                float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx+2]), ymax1[tx+1]), ymax3[tx+1]), data2[ptr0+1]), data2[ptr2+1]);
                maxv = fmaxf(maxv, d_Threshold[0]);
                float val = data2[ptr1+1];
                if (val<minv || val>maxv) {
                    float dxx = 2.0f*val - data2[ptr1+0] - data2[ptr1+2];
                    float dyy = 2.0f*val - data2[ptr0+1] - data2[ptr2+1];
                    float dxy = 0.25f*(data2[ptr2+2] + data2[ptr0+0] - data2[ptr0+2] - data2[ptr2+0]);
                    float tra = dxx + dyy;
                    float det = dxx*dyy - dxy*dxy;
                    if (tra*tra<d_EdgeLimit*det) {
                        float edge = __fdividef(tra*tra, det);
                        float dx = 0.5f*(data2[ptr1+2] - data2[ptr1+0]);
                        float dy = 0.5f*(data2[ptr2+1] - data2[ptr0+1]);
                        float ds = 0.5f*(data1[ptr1+1] - data3[ptr1+1]);
                        float dss = 2.0f*val - data3[ptr1+1] - data1[ptr1+1];
                        float dxs = 0.25f*(data3[ptr1+2] + data1[ptr1+0] - data1[ptr1+2] - data3[ptr1+0]);
                        float dys = 0.25f*(data3[ptr2+1] + data1[ptr0+1] - data3[ptr0+1] - data1[ptr2+1]);
                        float idxx = dyy*dss - dys*dys;
                        float idxy = dys*dxs - dxy*dss;
                        float idxs = dxy*dys - dyy*dxs;
                        float idyy = dxx*dss - dxs*dxs;
                        float idys = dxy*dxs - dxx*dys;
                        float idss = dxx*dyy - dxy*dxy;
                        float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
                        float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
                        float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
                        float pds = idet*(idxs*dx + idys*dy + idss*ds);
                        if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
                            pdx = __fdividef(dx, dxx);
                            pdy = __fdividef(dy, dyy);
                            pds = __fdividef(ds, dss);
                        }
                        float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
                        int maxPts = d_MaxNumPoints;
                        unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
                        idx = (idx>=maxPts ? maxPts-1 : idx);
                        d_Sift[idx + CUDASIFT_POINT_XPOS * maxPts] = xpos + pdx;
                        d_Sift[idx + CUDASIFT_POINT_YPOS * maxPts] = ypos - 1 + pdy;
                        d_Sift[idx + CUDASIFT_POINT_SCALE * maxPts] = d_Scales[0] * exp2f(pds * d_Factor);
                        d_Sift[idx + CUDASIFT_POINT_SHARPNESS * maxPts] = val + dval;
                        d_Sift[idx + CUDASIFT_POINT_EDGENESS * maxPts] = edge;
                        //printf("idx: %d %.1f %.1f %.2f\n", idx, d_Sift[idx + 0*maxPts], d_Sift[idx + 1*maxPts], edge);
                    }
                }
            }
        }
        __syncthreads();

        ptr0 = ptr1;
        ptr1 = ptr2;
        yq = (yq<2 ? yq+1 : 0);
    }
}

void cpu_compute_orientations(cv::cuda::GpuMat& image, cv::cuda::GpuMat& sift, int numPts, int maxPts)
{
    cv::cuda::PtrStepSzf gpu_image = image;
    cv::cuda::PtrStepSzf gpu_sift = sift;

    int p = gpu_image.step / sizeof(float);
    int h = gpu_image.rows;

    dim3 blocks(numPts);
    dim3 threads(32);
    gpu_compute_orientations<<<blocks, threads>>>(gpu_image.ptr(), gpu_sift.ptr(), maxPts, p, h);
    CUDA_SAFECALL(hipGetLastError());
    CUDA_SAFECALL(hipDeviceSynchronize());
}

__global__ void gpu_compute_orientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    for (int i=0;i<13;i++)
        hist[i*32+tx] = 0.0f;
    __syncthreads();
    float i2sigma2 = -1.0f/(2.0f*3.0f*3.0f);
    if (tx<15)
        gauss[tx] = exp(i2sigma2*(tx-7)*(tx-7));
    int xp = (int)(d_Sift[bx + CUDASIFT_POINT_XPOS * maxPts] - 6.5f);
    int yp = (int)(d_Sift[bx + CUDASIFT_POINT_YPOS * maxPts] - 6.5f);
    int px = xp & 15;
    int x = tx - px;

    for (int y=0;y<15;y++) {
        int memPos = 16*y + x;
        int xi = xp + x;
        int yi = yp + y;
        if (xi<0)
            xi = 0;
        if (xi>=w)
            xi = w-1;
        if (yi<0)
            yi = 0;
        if (yi>=h)
            yi = h-1;
        if (x>=0 && x<15)
            data[memPos] = g_Data[yi*w+xi];
    }
    __syncthreads();
    for (int y=1;y<14;y++) {
        int memPos = 16*y + x;
        if (x>=1 && x<14) {
            float dy = data[memPos+16] - data[memPos-16];
            float dx = data[memPos+1]  - data[memPos-1];
            int bin = 16.0f*atan2f(dy, dx)/3.1416f + 16.5f;
            if (bin==32)
                bin = 0;
            float grad = sqrtf(dx*dx + dy*dy);
            hist[32*(x-1)+bin] += grad*gauss[x]*gauss[y];
        }
    }

    __syncthreads();
    for (int y=0;y<5;y++)
        hist[y*32+tx] += hist[(y+8)*32+tx];

    __syncthreads();
    for (int y=0;y<4;y++)
        hist[y*32+tx] += hist[(y+4)*32+tx];

    __syncthreads();
    for (int y=0;y<2;y++)
        hist[y*32+tx] += hist[(y+2)*32+tx];

    __syncthreads();
    hist[tx] += hist[32+tx];

    __syncthreads();
    if (tx==0)
        hist[32] = 6*hist[0] + 4*(hist[1]+hist[31]) + (hist[2]+hist[30]);
    if (tx==1)
        hist[33] = 6*hist[1] + 4*(hist[2]+hist[0]) + (hist[3]+hist[31]);
    if (tx>=2 && tx<=29)
        hist[tx+32] = 6*hist[tx] + 4*(hist[tx+1]+hist[tx-1]) + (hist[tx+2]+hist[tx-2]);
    if (tx==30)
        hist[62] = 6*hist[30] + 4*(hist[31]+hist[29]) + (hist[0]+hist[28]);
    if (tx==31)
        hist[63] = 6*hist[31] + 4*(hist[0]+hist[30]) + (hist[1]+hist[29]);

    __syncthreads();
    float v = hist[32+tx];
    hist[tx] = (v>hist[32+((tx+1)&31)] && v>=hist[32+((tx+31)&31)] ? v : 0.0f);

    __syncthreads();
    if (tx==0) {
        float maxval1 = 0.0;
        float maxval2 = 0.0;
        int i1 = -1;
        int i2 = -1;
        for (int i=0;i<32;i++) {
            float v = hist[i];
            if (v>maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v>maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        float val1 = hist[32+((i1+1)&31)];
        float val2 = hist[32+((i1+31)&31)];
        float peak = i1 + 0.5f*(val1-val2) / (2.0f*maxval1-val1-val2);
        d_Sift[bx + CUDASIFT_POINT_ORIENTATION * maxPts] = 11.25f*(peak<0.0f ? peak+32.0f : peak);
        if (maxval2<0.8f*maxval1)
            i2 = -1;
        if (i2>=0) {
            float val1 = hist[32+((i2+1)&31)];
            float val2 = hist[32+((i2+31)&31)];
            float peak = i2 + 0.5f*(val1-val2) / (2.0f*maxval2-val1-val2);
            d_Sift[bx + CUDASIFT_POINT_SCORE * maxPts] = 11.25f*(peak<0.0f ? peak+32.0f : peak);
        } else {
            d_Sift[bx + CUDASIFT_POINT_SCORE * maxPts] = i2;
        }
    }
}

void cpu_subtract_multi(cv::cuda::GpuMat& dst, cv::cuda::GpuMat& src)
{
    assert(src.cols > 0 && src.rows > 0 && src.type() == CV_32FC1);

    dst = cv::cuda::GpuMat(src.size(), src.type());

    cv::cuda::PtrStepSzf gpu_src = src;
    cv::cuda::PtrStepSzf gpu_dst = dst;

    int w = gpu_dst.cols;
    int p = gpu_dst.step / sizeof(float);
    int h = gpu_dst.rows;

    dim3 blocks(iDivUp(w, SUBTRACTM_W), iDivUp(h, SUBTRACTM_H));
    dim3 threads(SUBTRACTM_W, SUBTRACTM_H, NUM_SCALES + 2);
    gpu_subtract_multi<<<blocks, threads>>>(gpu_dst.ptr(), gpu_src.ptr(), w, p, h);
    CUDA_SAFECALL(hipGetLastError());
    CUDA_SAFECALL(hipDeviceSynchronize());
}

__global__ void gpu_subtract_multi(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;
    int sz = height * pitch;
    int p = threadIdx.z * sz + y * pitch + x;
    if (x < width && y < height)
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    __syncthreads();
}

unsigned int cpu_find_points_multi(cv::cuda::GpuMat& src, cv::cuda::GpuMat& sift, float thresh, int maxPts, float edgeLimit, float scale, float factor, float lowestScale)
{
    assert(src.cols > 0 && src.rows > 0 && src.type() == CV_32FC1);

    cv::cuda::PtrStepSzf gpu_src = src;
    cv::cuda::PtrStepSzf gpu_sift = sift;

    int w = gpu_src.cols;
    int p = gpu_src.step / sizeof(float);
    int h = gpu_src.rows;

    float scales[NUM_SCALES];
    int nScales = 0;
    float diffScale = pow(2.0f, factor);
    for (int i=0;i<NUM_SCALES;i++) {
        if (scale>=lowestScale)
            scales[nScales++] = scale;
        scale *= diffScale;
    }

    CUDA_SAFECALL(set_threshold(thresh, -thresh));
    CUDA_SAFECALL(set_edge_limit(edgeLimit));
    CUDA_SAFECALL(set_scales(scales, NUM_SCALES));
    CUDA_SAFECALL(set_factor(factor));
    CUDA_SAFECALL(set_max_points(maxPts));

    CUDA_SAFECALL(set_point_counter(0));

    if (nScales > 0) {
        dim3 blocks(iDivUp(w, MINMAX_W) * nScales, iDivUp(h, MINMAX_H));
        dim3 threads(MINMAX_W + 2);
        gpu_find_points_multi<<<blocks, threads>>>(gpu_src.ptr(), gpu_sift.ptr(), w, p, h, nScales);
        CUDA_SAFECALL(hipGetLastError());
        CUDA_SAFECALL(hipDeviceSynchronize());
    }

    unsigned int total_points = 0;
    CUDA_SAFECALL(get_point_counter(total_points));
    return total_points;
}

__global__ void gpu_find_points_multi(float *d_Data0, float *d_Sift, int width, int pitch, int height, int nScales)
{
    #define MEMWID (MINMAX_W + 2)
    __shared__ float data1[3*MEMWID], data2[3*MEMWID], data3[3*MEMWID];
    __shared__ float ymin1[MEMWID],   ymin2[MEMWID],   ymin3[MEMWID];
    __shared__ float ymax1[MEMWID],   ymax2[MEMWID],   ymax3[MEMWID];

    const int tx = threadIdx.x;
    const int block = blockIdx.x/nScales;
    const int scale = blockIdx.x - nScales*block;
    const int minx = block*MINMAX_W;
    const int maxx = min(minx + MINMAX_W, width);
    const int xpos = minx + tx;
    const int size = pitch*height;
    const float *d_Data1 = d_Data0 + size*scale;
    const float *d_Data2 = d_Data1 + size;
    const float *d_Data3 = d_Data2 + size;
    //if (block==0 && blockIdx.y==0 && tx==0)
    //  printf("%08x %08x %08x %d %d\n", d_Data1, d_Data2, d_Data3, pitch, height);

    int ptr0 = tx;
    int ptr1 = tx;
    int yq = 0;
    for (int y=0;y<MINMAX_H+2;y++) {

        int xposr = xpos - 1;
        int ypos = MINMAX_H*blockIdx.y + y - 1;
        int yptr = min(max(ypos, 0), height - 1)*pitch;

        int ptr2 = yq*MEMWID + tx;
        if (xposr<0) {
            data1[ptr2] = 0;
            data2[ptr2] = 0;
            data3[ptr2] = 0;
        } else if (xposr>=width) {
            data1[ptr2] = 0;
            data2[ptr2] = 0;
            data3[ptr2] = 0;
        } else {
            data1[ptr2] = d_Data1[yptr + xposr];
            data2[ptr2] = d_Data2[yptr + xposr];
            data3[ptr2] = d_Data3[yptr + xposr];
        }
        //__syncthreads();
        if (y>1) {
            float min1 = fminf(fminf(data1[ptr0], data1[ptr1]), data1[ptr2]);
            float min2 = fminf(fminf(data2[ptr0], data2[ptr1]), data2[ptr2]);
            float min3 = fminf(fminf(data3[ptr0], data3[ptr1]), data3[ptr2]);
            float max1 = fmaxf(fmaxf(data1[ptr0], data1[ptr1]), data1[ptr2]);
            float max2 = fmaxf(fmaxf(data2[ptr0], data2[ptr1]), data2[ptr2]);
            float max3 = fmaxf(fmaxf(data3[ptr0], data3[ptr1]), data3[ptr2]);
            ymin1[tx] = min1;
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymin3[tx] = min3;
            ymax1[tx] = max1;
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
            ymax3[tx] = max3;
        }
        //__syncthreads();
        if (y>1) {
            if (tx<MINMAX_W && xpos<maxx) {
                float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx+2]), ymin1[tx+1]), ymin3[tx+1]), data2[ptr0+1]), data2[ptr2+1]);
                minv = fminf(minv, d_Threshold[1]);
                float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx+2]), ymax1[tx+1]), ymax3[tx+1]), data2[ptr0+1]), data2[ptr2+1]);
                maxv = fmaxf(maxv, d_Threshold[0]);
                float val = data2[ptr1+1];
                if (val<minv || val>maxv) {
                    float dxx = 2.0f*val - data2[ptr1+0] - data2[ptr1+2];
                    float dyy = 2.0f*val - data2[ptr0+1] - data2[ptr2+1];
                    float dxy = 0.25f*(data2[ptr2+2] + data2[ptr0+0] - data2[ptr0+2] - data2[ptr2+0]);
                    float tra = dxx + dyy;
                    float det = dxx*dyy - dxy*dxy;
                    if (tra*tra<d_EdgeLimit*det) {
                        float edge = __fdividef(tra*tra, det);
                        float dx = 0.5f*(data2[ptr1+2] - data2[ptr1+0]);
                        float dy = 0.5f*(data2[ptr2+1] - data2[ptr0+1]);
                        float ds = 0.5f*(data1[ptr1+1] - data3[ptr1+1]);
                        float dss = 2.0f*val - data3[ptr1+1] - data1[ptr1+1];
                        float dxs = 0.25f*(data3[ptr1+2] + data1[ptr1+0] - data1[ptr1+2] - data3[ptr1+0]);
                        float dys = 0.25f*(data3[ptr2+1] + data1[ptr0+1] - data3[ptr0+1] - data1[ptr2+1]);
                        float idxx = dyy*dss - dys*dys;
                        float idxy = dys*dxs - dxy*dss;
                        float idxs = dxy*dys - dyy*dxs;
                        float idyy = dxx*dss - dxs*dxs;
                        float idys = dxy*dxs - dxx*dys;
                        float idss = dxx*dyy - dxy*dxy;
                        float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
                        float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
                        float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
                        float pds = idet*(idxs*dx + idys*dy + idss*ds);
                        if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
                            pdx = __fdividef(dx, dxx);
                            pdy = __fdividef(dy, dyy);
                            pds = __fdividef(ds, dss);
                        }
                        float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
                        int maxPts = d_MaxNumPoints;
                        unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
                        idx = (idx>=maxPts ? maxPts-1 : idx);
                        d_Sift[idx + CUDASIFT_POINT_XPOS * maxPts] = xpos + pdx;
                        d_Sift[idx + CUDASIFT_POINT_YPOS * maxPts] = ypos - 1 + pdy;
                        d_Sift[idx + CUDASIFT_POINT_SCALE * maxPts] = d_Scales[scale] * exp2f(pds*d_Factor);
                        d_Sift[idx + CUDASIFT_POINT_SHARPNESS * maxPts] = val + dval;
                        d_Sift[idx + CUDASIFT_POINT_EDGENESS * maxPts] = edge;
                    }
                }
            }
        }
        __syncthreads();
        ptr0 = ptr1;
        ptr1 = ptr2;
        yq = (yq<2 ? yq+1 : 0);
    }
}

#define RADIUS 4

void cpu_lowpass(cv::cuda::GpuMat& dst, cv::cuda::GpuMat& src, float baseBlur, float diffScale, float initBlur)
{
    // Checks
    assert(src.cols > 0 && src.rows > 0 && src.type() == CV_32FC1);

    // Matrices
    dst = cv::cuda::GpuMat(src.size(), src.type());
    cv::cuda::GpuMat tmp(src.size(), src.type());

    // Pointers
    cv::cuda::PtrStepSzf gpu_src = src;
    cv::cuda::PtrStepSzf gpu_dst = dst;
    cv::cuda::PtrStepSzf gpu_tmp = tmp;

    // Convolution kernel
    float kernel[12 * 16];
    float scale = baseBlur;
    for (int i = 0; i < NUM_SCALES + 3;i++) {
        float kernelSum = 0.0f;
        float var = scale*scale - initBlur*initBlur;
        for (int j=-RADIUS;j<=RADIUS;j++) {
            kernel[16 * i + j + RADIUS] = (float)expf(-(double)j*j/2.0/var);
            kernelSum += kernel[16*i+j+RADIUS];
        }
        for (int j=-RADIUS;j<=RADIUS;j++)
            kernel[16*i+j+RADIUS] /= kernelSum;
        scale *= diffScale;
    }
    CUDA_SAFECALL(set_kernel(kernel, 12 * 16));

    int width = gpu_src.cols;
    int pitch = gpu_src.step / sizeof(float);
    int height = gpu_src.rows;

    dim3 block_rows(iDivUp(width, CONVROW_W) * (NUM_SCALES + 3), height);
    dim3 thread_rows(CONVROW_W + 2 * RADIUS);
    gpu_lowpass_row<<<block_rows, thread_rows>>>(gpu_tmp.ptr(), gpu_src.ptr(), width, pitch, height);
    CUDA_SAFECALL(hipGetLastError());
    CUDA_SAFECALL(hipDeviceSynchronize());

    dim3 block_cols(iDivUp(width, CONVCOL_W) * (NUM_SCALES + 3), iDivUp(height, CONVCOL_H));
    dim3 thread_cols(CONVCOL_W, CONVCOL_S);
    gpu_lowpass_col<<<block_cols, thread_cols>>>(gpu_dst.ptr(), gpu_tmp.ptr(), width, pitch, height);
    CUDA_SAFECALL(hipGetLastError());
    CUDA_SAFECALL(hipDeviceSynchronize());
}

__global__ void gpu_lowpass_row(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVROW_W + 2 * RADIUS];
    const int tx = threadIdx.x;
    const int block = blockIdx.x / (NUM_SCALES + 3);
    const int scale = blockIdx.x - (NUM_SCALES + 3) * block;
    const int xout = block * CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y * pitch;
    const int writePos = yptr + height * pitch * scale + xout;
    float *kernel = d_Kernel + scale * 16;

    if (loadPos < 0)
        data[tx] = d_Data[yptr];
    else if (loadPos >= width)
        data[tx] = d_Data[yptr + width - 1];
    else
        data[tx] = d_Data[yptr + loadPos];

    __syncthreads();

    if (xout < width && tx < CONVROW_W) {
        d_Result[writePos] =
            (data[tx+0] + data[tx+8])*kernel[0] +
            (data[tx+1] + data[tx+7])*kernel[1] +
            (data[tx+2] + data[tx+6])*kernel[2] +
            (data[tx+3] + data[tx+5])*kernel[3] +
            data[tx+4]*kernel[4];
    }

    __syncthreads();
}

__global__ void gpu_lowpass_col(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVCOL_W * (CONVCOL_H + 2 * RADIUS)];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int block = blockIdx.x/(NUM_SCALES+3);
    const int scale = blockIdx.x - (NUM_SCALES+3)*block;
    const int miny = blockIdx.y*CONVCOL_H;
    const int maxy = min(miny + CONVCOL_H, height) - 1;
    const int totStart = miny - RADIUS;
    const int totEnd = maxy + RADIUS;
    const int colStart = block*CONVCOL_W + tx;
    const int colEnd = colStart + (height-1)*pitch;
    const int sStep = CONVCOL_W * CONVCOL_S;
    const int gStep = pitch * CONVCOL_S;
    float *kernel = d_Kernel + scale * 16;
    const int size = pitch * height * scale;

    d_Result += size;
    d_Data += size;

    if (colStart < width) {
        float *sdata = data + ty * CONVCOL_W + tx;
        int gPos = colStart + (totStart + ty) * pitch;
        for (int y = totStart + ty; y <= totEnd; y += blockDim.y) {
            if (y < 0)
                sdata[0] = d_Data[colStart];
            else if (y >= height)
                sdata[0] = d_Data[colEnd];
            else
                sdata[0] = d_Data[gPos];
            sdata += sStep;
            gPos += gStep;
        }
    }

    __syncthreads();

    if (colStart < width) {
        float* sdata = data + ty * CONVCOL_W + tx;
        int gPos = colStart + (miny + ty) * pitch;
        for (int y = miny + ty; y <= maxy; y += blockDim.y) {
            d_Result[gPos] =
                (sdata[0 * CONVCOL_W] + sdata[8 * CONVCOL_W]) * kernel[0] +
                (sdata[1 * CONVCOL_W] + sdata[7 * CONVCOL_W]) * kernel[1] +
                (sdata[2 * CONVCOL_W] + sdata[6 * CONVCOL_W]) * kernel[2] +
                (sdata[3 * CONVCOL_W] + sdata[5 * CONVCOL_W]) * kernel[3] +
                sdata[4 * CONVCOL_W] * kernel[4];
            sdata += sStep;
            gPos += gStep;
        }
    }
}

};
